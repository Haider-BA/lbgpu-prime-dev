//##############################################################################
//
// lbgpu_prime.cu
//
//  - Lattice Boltzmann
//
//  - D2Q9 / D3Q19
//
#define DP_ON 0

#if DP_ON
typedef double real; // To be relocated (in flags.in?).
#else
typedef float real; // To be relocated (in flags.in?).
#endif

#ifdef __HIPCC__
real* f_mem_d;
int f_mem_size;
real* mv_mem_d;
int mv_mem_size;
unsigned char* solids_mem_d;
int solids_mem_size;
int* is_end_of_frame_mem_d;
#endif

#include "lbgpu_prime.h"
int main( int argc, char **argv)
{
  int subs;
  int time, frame;

  struct lattice_struct *lattice;

  setbuf( stdout, (char*)NULL); // Don't buffer screen output.

#if VERBOSITY_LEVEL > 0
  printf("%s %d >> lbgpu_prime.c: main() -- Hello.\n", __FILE__, __LINE__);
#endif /* VERBOSITY_LEVEL > 0 */

  construct_lattice( &lattice, argc, argv);

  process_tic( lattice);

  init_problem( lattice);

  set_time( lattice, 0);
  set_frame( lattice, 0);

  output_frame( lattice);

  process_toc( lattice);
  display_etime( lattice);
  process_tic( lattice);

#ifdef __HIPCC__

  if( get_LX( lattice) % get_BX( lattice) != 0)
  {
    printf("\n%s %d ERROR: LX = %d must be divisible by BX = %d. (Exiting.)\n\n"
        , __FILE__
        , __LINE__
        , get_LX( lattice)
        , get_BX( lattice));
    process_exit(1);
  }
  if( get_LY( lattice) % get_BY( lattice) != 0)
  {
    printf("\n%s %d ERROR: LY = %d must be divisible by BY = %d. (Exiting.)\n\n"
        , __FILE__
        , __LINE__
        , get_LY( lattice)
        , get_BY( lattice));
    process_exit(1);
  }
  if( get_LZ( lattice) % get_BZ( lattice) != 0)
  {
    printf("\n%s %d ERROR: LX = %d must be divisible by BX = %d. (Exiting.)\n\n"
        , __FILE__
        , __LINE__
        , get_LZ( lattice)
        , get_BZ( lattice));
    process_exit(1);
  }


  dim3 blockDim( get_BX( lattice), get_BY( lattice), get_BZ( lattice));
  dim3 gridDim( get_LX( lattice) / get_BX( lattice),
      get_LY( lattice) / get_BY( lattice),
      get_LZ( lattice) / get_BZ( lattice) );

#endif

#ifdef __HIPCC__

  hipMemcpy( solids_mem_d + get_EndBoundSize( lattice)
      , get_solids_ptr(lattice, 0)
      , get_NumNodes( lattice)*sizeof(unsigned char)
      , hipMemcpyHostToDevice);

  checkCUDAError( __FILE__, __LINE__, "hipMemcpy");

  // Do boundary swap for solids.  This only needs to be done
  // once, of course.

#if PARALLEL
  // We'll want something involving the MPI buffers here.

  solid_send_recv_begin( lattice, 0);
  solid_send_recv_end( lattice, 0);

  // North / Top end
  hipMemcpy( solids_mem_d + get_EndBoundSize( lattice) + get_NumNodes( lattice)
      , lattice->process.neg_dir_solid_to_recv
      , get_EndBoundSize( lattice)*sizeof(unsigned char)
      , hipMemcpyHostToDevice);

  checkCUDAError( __FILE__, __LINE__, "solid swap");

  // South / Bottom end
  hipMemcpy( solids_mem_d
      , lattice->process.pos_dir_solid_to_recv
      , get_EndBoundSize( lattice)*sizeof(unsigned char)
      , hipMemcpyHostToDevice);

  checkCUDAError( __FILE__, __LINE__, "solid swap");

#else
  // North / Top end
  hipMemcpy( solids_mem_d + get_EndBoundSize( lattice) + get_NumNodes( lattice)
      , solids_mem_d + get_EndBoundSize( lattice)
      , get_EndBoundSize( lattice)*sizeof(unsigned char)
      , hipMemcpyDeviceToDevice);

  checkCUDAError( __FILE__, __LINE__, "solid swap");

  // South / Bottom end
  hipMemcpy( solids_mem_d
      , solids_mem_d + get_NumNodes( lattice)
      , get_EndBoundSize( lattice)*sizeof(unsigned char)
      , hipMemcpyDeviceToDevice);

  checkCUDAError( __FILE__, __LINE__, "solid swap");
#endif

  int dir;
  const int ubdir = get_NumVelDirs( lattice) 
    - get_NumBoundDirs( lattice);

  for( subs = 0; subs < get_NumSubs( lattice); subs++)
  {
    // In the final version, the CPU arrays may well have
    // the same 'additional boundary' structure as the GPU
    // arrays, in which case a single memcpy can be performed.
    hipMemcpy( f_mem_d + subs * cumul_stride[get_NumVelDirs( lattice)]
        , get_fptr(lattice, subs, 0,0,0, 0,0,0, 0)
        , get_NumNodes( lattice)
        *(ubdir)
        *sizeof(real)
        , hipMemcpyHostToDevice);

    checkCUDAError( __FILE__, __LINE__, "hipMemcpy");

    for( dir=ubdir; dir < get_NumVelDirs( lattice); dir+=2)
    {

      hipMemcpy( f_mem_d + cumul_stride[dir]
          + subs * cumul_stride[get_NumVelDirs( lattice)]
          , get_fptr(lattice, subs, 0,0,0, 0,0,0, dir)
          , 2*get_NumNodes( lattice)
          *sizeof(real)
          , hipMemcpyHostToDevice);

      checkCUDAError( __FILE__, __LINE__, "hipMemcpy");
    }


    hipMemcpy( mv_mem_d + subs*get_NumNodes( lattice)*(1 + get_NumDims( lattice))
        , get_rho_ptr(lattice, subs, 0)
        , (1. + get_NumDims( lattice))
        *get_NumNodes( lattice)
        *sizeof(real)
        , hipMemcpyHostToDevice);

    checkCUDAError( __FILE__, __LINE__, "hipMemcpy");

  }

  int temp = 0;
  hipMemcpyToSymbol(HIP_SYMBOL( is_end_of_frame_mem_c), &temp, sizeof(int));
#if 0
  hipMemcpy( is_end_of_frame_mem_d
      , &temp
      , sizeof(int)
      , hipMemcpyHostToDevice);

  checkCUDAError( __FILE__, __LINE__, "hipMemcpy");
#endif
#if TEXTURE_FETCH
  hipBindTexture(0, tex_solid, solids_mem_d);
#endif

#endif

  for( frame = 0, time=1; time<=get_NumTimeSteps( lattice); time++)
  {
    set_time( lattice, time);

    // Do boundary swaps. The direction depends on
    // whether time is even or odd. Here, time is odd.
#if PARALLEL
    for( subs = 0; subs < get_NumSubs( lattice); subs++)
    {
#ifdef __HIPCC__
      for( dir=ubdir; dir < get_NumVelDirs( lattice); dir++)
      {
        pdf_boundary_parallel( lattice, f_mem_d, cumul_stride
            , subs, dir, time, DEVICE_TO_HOST);
      }
#endif

      process_send_recv_begin( lattice, subs);
      process_send_recv_end(lattice, subs);

#ifdef __HIPCC__
      for( dir=ubdir; dir < get_NumVelDirs( lattice); dir++)
      {
        pdf_boundary_parallel( lattice, f_mem_d, cumul_stride
            , subs, dir, time, HOST_TO_DEVICE);
      }
#endif
    }
#else
#ifdef __HIPCC__
    for( subs = 0; subs < get_NumSubs( lattice); subs++)
    {
      for( dir=ubdir; dir < get_NumVelDirs( lattice); dir++)
      {
        pdf_boundary_swap( lattice, f_mem_d, cumul_stride
            , subs, dir, time);
      }
    }
#endif
#endif


    // Time steps are combined in a somewhat awkward way in order to minimize
    // the amount of temporary storage required and to allow the nodes to
    // be updated independently of neighbor nodes. This facilitates an
    // efficient GPU implementation.
    //
    // c.f., Bailey, Myre, Walsh et. al., Accelerating Lattice Boltzmann Fluid
    // Flow Simulations using Graphics Processors, ICPP 2009.
    // http://www.tc.umn.edu/~bail0253/

      //XXXXXXX
#if 1
#ifdef __HIPCC__
    k_stream_collide_stream
      <<<
      gridDim
      , blockDim
      , sizeof(real)*( get_NumVelDirs( lattice)
          + get_NumDims( lattice)+1 )
      * get_BX( lattice)
      * get_BY( lattice)
      * get_BZ( lattice)
      >>>( f_mem_d, mv_mem_d, solids_mem_d);
    hipDeviceSynchronize();
    checkCUDAError( __FILE__, __LINE__, "k_stream_collide_stream");

#else
    stream_collide_stream( lattice);
#endif
#endif
    set_time( lattice, ++time);
    //XXXXXX
    // Do boundary swaps. The direction depends on
    // whether time is even or odd. Here, time is odd.
#if PARALLEL
    for( subs = 0; subs < get_NumSubs( lattice); subs++)
    {
#ifdef __HIPCC__
      for( dir=ubdir; dir < get_NumVelDirs( lattice); dir++)
      {
        pdf_boundary_parallel( lattice, f_mem_d, cumul_stride
            , subs, dir, time, DEVICE_TO_HOST);
      }
#endif

      process_send_recv_begin( lattice, subs);
      process_send_recv_end(lattice, subs);

#ifdef __HIPCC__
      for( dir=ubdir; dir < get_NumVelDirs( lattice); dir++)
      {
        pdf_boundary_parallel( lattice, f_mem_d, cumul_stride
            , subs, dir, time, HOST_TO_DEVICE);
      }
#endif
    }
#else
#ifdef __HIPCC__
    for( subs = 0; subs < get_NumSubs( lattice); subs++)
    {
      for( dir=ubdir; dir < get_NumVelDirs( lattice); dir++)
      {
        pdf_boundary_swap( lattice, f_mem_d, cumul_stride
            , subs, dir, time);
      }
    }
#endif
#endif

#ifdef __HIPCC__
#if 1
    if( is_end_of_frame(lattice,time))
    {
      temp = 1;
      hipMemcpyToSymbol(HIP_SYMBOL( is_end_of_frame_mem_c), &temp, sizeof(int));
#if 0
      hipMemcpy( is_end_of_frame_mem_d
          , &temp
          , sizeof(int)
          , hipMemcpyHostToDevice);

      checkCUDAError( __FILE__, __LINE__, "hipMemcpy");
#endif
    }
#endif
#if 1
    k_collide
      <<<
      gridDim
      , blockDim
      , sizeof(real)*( get_NumVelDirs( lattice)
          + get_NumDims( lattice)+1 )
      * get_BX( lattice)
      * get_BY( lattice)
      * get_BZ( lattice)
      >>>( f_mem_d, mv_mem_d, solids_mem_d);
    hipDeviceSynchronize();
    checkCUDAError( __FILE__, __LINE__, "k_collide");
#endif
#if 1
    if( is_end_of_frame(lattice,time))
    {
      temp = 0;
      hipMemcpyToSymbol(HIP_SYMBOL( is_end_of_frame_mem_c), &temp, sizeof(int));
#if 0
      hipMemcpy( is_end_of_frame_mem_d
          , &temp
          , sizeof(int)
          , hipMemcpyHostToDevice);
      checkCUDAError( __FILE__, __LINE__, "hipMemcpy");
#endif
    }
#endif
#else
    collide( lattice);
#endif
    if( is_end_of_frame(lattice,time))
    {
      set_frame( lattice, ++frame);

#ifdef __HIPCC__
      for( subs = 0; subs < get_NumSubs( lattice); subs++)
      {
        printf("Transferring subs %d "
            "macrovars from device to host for output. \n", subs);
        hipMemcpy( get_rho_ptr(lattice, subs, 0)
            , mv_mem_d
            + subs*get_NumNodes( lattice)*( 1 + get_NumDims( lattice))
            + 0*get_NumNodes( lattice)
            , get_NumNodes( lattice)*sizeof(real)
            , hipMemcpyDeviceToHost);
        checkCUDAError( __FILE__, __LINE__, "hipMemcpy");

        hipMemcpy( get_ux_ptr(lattice, subs, 0)
            , mv_mem_d
            + subs*get_NumNodes( lattice)*( 1 + get_NumDims( lattice))
            + 1*get_NumNodes( lattice)
            , get_NumNodes( lattice)*sizeof(real)
            , hipMemcpyDeviceToHost);
        checkCUDAError( __FILE__, __LINE__, "hipMemcpy");

        hipMemcpy( get_uy_ptr(lattice, subs, 0)
            , mv_mem_d
            + subs*get_NumNodes( lattice)*( 1 + get_NumDims( lattice))
            + 2*get_NumNodes( lattice)
            , get_NumNodes( lattice)*sizeof(real)
            , hipMemcpyDeviceToHost);
        checkCUDAError( __FILE__, __LINE__, "hipMemcpy");

        if(get_NumDims( lattice) == 3)
        {
          hipMemcpy( get_uz_ptr(lattice, subs, 0)
              , mv_mem_d
              + subs*get_NumNodes( lattice)*( 1 + get_NumDims( lattice))
              + 3*get_NumNodes( lattice)
              , get_NumNodes( lattice)*sizeof(real)
              , hipMemcpyDeviceToHost);
          checkCUDAError( __FILE__, __LINE__, "hipMemcpy");

        }
        if( do_write_f_txt_file( lattice))
        {  
          // In the final version, the CPU arrays may well have
          // the same 'additional boundary' structure as the GPU
          // arrays, in which case a single memcpy can be performed.
          hipMemcpy( get_fptr(lattice, subs, 0,0,0, 0,0,0, 0)
              , f_mem_d + subs * cumul_stride[get_NumVelDirs( lattice)]
              , get_NumNodes( lattice)
              *(ubdir)
              *sizeof(real)
              , hipMemcpyDeviceToHost);

          checkCUDAError( __FILE__, __LINE__, "hipMemcpy");

          for( dir=ubdir; dir < get_NumVelDirs( lattice); dir+=2)
          {

            hipMemcpy( get_fptr(lattice, subs, 0,0,0, 0,0,0, dir)
                , f_mem_d + cumul_stride[dir]
                + subs * cumul_stride[get_NumVelDirs( lattice)]
                , 2*get_NumNodes( lattice)
                *sizeof(real)
                , hipMemcpyDeviceToHost);

            checkCUDAError( __FILE__, __LINE__, "hipMemcpy");
          }
        }
      }
#endif

      output_frame( lattice);
      process_toc( lattice);
      display_etime( lattice);

#ifdef __HIPCC__
      printf("**************\n");
      printf("  __HIPCC__\n");
      printf("**************\n");
#endif
    }

  } /* for( time=1; time<=lattice->NumTimeSteps; time++) */

  process_barrier();
  process_toc( lattice);
  display_etime( lattice);

  destruct_lattice( lattice);

#if VERBOSITY_LEVEL > 0
  printf("\n");
  printf("%s %d >> lbgpu_prime.c: main() -- Terminating normally.\n",
      __FILE__, __LINE__);
  printf("\n");
#endif /* VERBOSITY_LEVEL > 0 */

  return 0;

} /* int main( int argc, char **argv) */
